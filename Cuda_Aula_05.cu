#include "hip/hip_runtime.h"
// CUDA Image Blur Filter

#include <iostream>
#include <string>
#include <vector>
#include <cmath>
#include <chrono>

#include <chrono> // For timing CPU operations

#include "hip/hip_runtime.h"
#include ""

// Define these before including stb_image.h to act as the implementation
#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"

// Macro to wrap CUDA API calls and check for errors
#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ \
                  << ": " << hipGetErrorString(err_) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

// CUDA Kernel for a Box Blur filter
// Each thread processes one pixel of the output image.
__global__ void blurKernel(const unsigned char* in, unsigned char* out, int width, int height, int channels, int radius) {
    // Calculate the global x and y coordinates of the pixel for this thread
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check: ensure the thread is within the image dimensions
    if (col < width && row < height) {
        // Accumulators for the color channels (Red, Green, Blue)
        float r_acc = 0.0f, g_acc = 0.0f, b_acc = 0.0f;
        int pixel_count = 0;

        // Iterate over the neighborhood defined by the blur radius
        for (int y = -radius; y <= radius; ++y) {
            for (int x = -radius; x <= radius; ++x) {
                int current_row = row + y;
                int current_col = col + x;

                // Boundary check for the neighborhood pixels
                if (current_row >= 0 && current_row < height && current_col >= 0 && current_col < width) {
                    // Calculate the index of the neighboring pixel in the 1D array
                    int idx = (current_row * width + current_col) * channels;

                    // Accumulate color values
                    r_acc += in[idx + 0];
                    g_acc += in[idx + 1];
                    b_acc += in[idx + 2];
                    // Alpha channel (if present) is ignored for the blur calculation

                    pixel_count++;
                }
            }
        }

        // Calculate the average color value
        int out_idx = (row * width + col) * channels;
        out[out_idx + 0] = static_cast<unsigned char>(r_acc / pixel_count);
        out[out_idx + 1] = static_cast<unsigned char>(g_acc / pixel_count);
        out[out_idx + 2] = static_cast<unsigned char>(b_acc / pixel_count);

        // If there's an alpha channel, copy it directly
        if (channels == 4) {
            out[out_idx + 3] = in[out_idx + 3];
        }
    }
}


int main(int argc, char** argv) {
    // --- 1. Setup and Initialization ---
    const std::string input_filename = "Images/apple.jpg";
    const std::string output_filename = "Images/output.jpg";
    const int blur_radius = 5; // The radius of the blur kernel

    // CUDA events for timing GPU operations
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;

    // --- 2. Load Image and Allocate Host (CPU) Memory ---
    std::cout << "Loading image and allocating host memory..." << std::endl;
    auto start_cpu_alloc = std::chrono::high_resolution_clock::now();

    int width, height, channels;
    // Load image from file. The '0' forces stb_image to decide the number of channels.
    unsigned char* h_img_in = stbi_load(input_filename.c_str(), &width, &height, &channels, 0);
    if (h_img_in == nullptr) {
        std::cerr << "Error: Could not load image '" << input_filename << "'.\n";
        std::cerr << "Please ensure the image exists in the same directory as the executable." << std::endl;
        return 1;
    }
    std::cout << "-> Loaded '" << input_filename << "' (" << width << "x" << height << ", " << channels << " channels)" << std::endl;

    // Allocate host memory for the output image
    size_t img_size = width * height * channels * sizeof(unsigned char);
    unsigned char* h_img_out = (unsigned char*)malloc(img_size);

    auto stop_cpu_alloc = std::chrono::high_resolution_clock::now();
    auto duration_cpu_alloc = std::chrono::duration_cast<std::chrono::microseconds>(stop_cpu_alloc - start_cpu_alloc);
    std::cout << "-> Host allocation and image load time: " << duration_cpu_alloc.count() << " microseconds\n" << std::endl;

    // --- 3. Allocate Device (GPU) Memory ---
    std::cout << "Allocating device memory..." << std::endl;
    CUDA_CHECK(hipEventRecord(start));

    unsigned char* d_img_in = nullptr, * d_img_out = nullptr;
    CUDA_CHECK(hipMalloc(&d_img_in, img_size));
    CUDA_CHECK(hipMalloc(&d_img_out, img_size));

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "-> Device allocation time: " << milliseconds << " ms\n" << std::endl;

    // --- 4. Copy Data from Host to Device ---
    std::cout << "Copying image from host to device..." << std::endl;
    CUDA_CHECK(hipEventRecord(start));

    CUDA_CHECK(hipMemcpy(d_img_in, h_img_in, img_size, hipMemcpyHostToDevice));

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "-> Host to Device copy time: " << milliseconds << " ms\n" << std::endl;

    // --- 5. Launch the CUDA Kernel ---
    std::cout << "Launching CUDA blur kernel..." << std::endl;

    // Define thread block and grid dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    CUDA_CHECK(hipEventRecord(start));

    blurKernel << <gridDim, blockDim >> > (d_img_in, d_img_out, width, height, channels, blur_radius);

    CUDA_CHECK(hipGetLastError()); // Check for errors during kernel launch

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "-> Kernel execution time: " << milliseconds << " ms\n" << std::endl;

    // --- 6. Copy Results from Device to Host ---
    std::cout << "Copying blurred image from device to host..." << std::endl;
    CUDA_CHECK(hipEventRecord(start));

    CUDA_CHECK(hipMemcpy(h_img_out, d_img_out, img_size, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "-> Device to Host copy time: " << milliseconds << " ms\n" << std::endl;

    // --- 7. Save Image and Free Memory ---
    std::cout << "Saving blurred image..." << std::endl;
    stbi_write_jpg(output_filename.c_str(), width, height, channels, h_img_out, 100); // 100 is quality
    std::cout << "-> Saved '" << output_filename << "'" << std::endl;

    std::cout << "\nFreeing memory..." << std::endl;
    // Free host memory
    stbi_image_free(h_img_in);
    free(h_img_out);

    // Free device memory
    CUDA_CHECK(hipFree(d_img_in));
    CUDA_CHECK(hipFree(d_img_out));

    // Destroy CUDA events
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}